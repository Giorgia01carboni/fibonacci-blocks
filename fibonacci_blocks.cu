
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Check if a number is a Fibonacci number
 */
__device__ bool isFib(int s) {
    int a = 0, b = 1, c = 0;

    if (s == 0 || s == 1) return true;

    while (c < s) {
        c = a + b;
        a = b;
        b = c;
    }
    return c == s;
}

/*
 * Show DIMs & IDs for grid, block and thread
 */
__global__ void checkIndex(void) {

  int s = threadIdx.x * blockDim.x + threadIdx.y * blockDim.y + blockIdx.x + blockIdx.y;
  
  if (isFib(s)) {
      printf("threadIdx: (%d, %d)  blockIdx: (%d, %d) \nblockDim: (%d, %d)  gridDim: (%d, %d)\n\n", 
             threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, 
             blockDim.x, blockDim.y, gridDim.x, gridDim.y);
  }
}

int main(int argc, char **argv) {

    // grid and block structure
    dim3 block(7, 6);
    dim3 grid(2, 2);

    // Print from host
    printf("Print from host:\n");
    printf("grid.x = %d\t grid.y = %d\t grid.z = %d\n", grid.x, grid.y, grid.z);
    printf("block.x = %d\t block.y = %d\t block.z %d\n\n", block.x, block.y, block.z);

    // Print from device
    printf("Print from device:\n");
    checkIndex<<<grid, block>>>();

    // Ensure all threads complete
    hipDeviceSynchronize();

    // reset device
    hipDeviceReset();
    return 0;
}
